#include "hip/hip_runtime.h"
#include "../include/utils.h"
#include <hip/hip_runtime.h>

#define NUM_RUNS 10
#define BLOCK_WIDTH 32
#define BLOCK_WIDTH_OPT 8

#define CUDA_CHECK(func)                                                     	   \
	do {                                                                           \
		hipError_t status = (func);                                               \
		if (status != hipSuccess) {                                               \
			printf("CUDA API failed at line %d with error: %s (%d)\n", __LINE__,   \
				hipGetErrorString(status), status);                               \
			exit(EXIT_FAILURE);                                                    \
		}                                                                          \
	} while (0)

#define CHECK(name) \
	float *d_Aref_ ## name, *d_Bref_ ## name, *d_Cref_ ## name; \
	std::cerr << "checking " << #name << std::endl; \
	CUDA_CHECK(hipMalloc(&d_Aref_ ## name, Ref::M * Ref::K * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_Bref_ ## name, Ref::K * Ref::N * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_Cref_ ## name, Ref::M * Ref::N * sizeof(float))); \
	CUDA_CHECK(hipMemcpy(d_Aref_ ## name, ref.A, Ref::M * Ref::K * sizeof(float), hipMemcpyHostToDevice)); \
	CUDA_CHECK(hipMemcpy(d_Bref_ ## name, ref.B, Ref::K * Ref::N * sizeof(float), hipMemcpyHostToDevice)); \
	float* d_Cref_INI_ ## name = new float[M * N](); \
	for (int i = 0; i < Ref::M; i++) { \
		for (int j = 0; j < Ref::N; j++) { \
			d_Cref_INI_ ## name[i * Ref::N + j] = 0; \
		} \
	} \
	CUDA_CHECK(hipMemcpy(d_Cref_ ## name, d_Cref_INI_ ## name, Ref::M * Ref::N * sizeof(float), hipMemcpyHostToDevice)); \
	name(d_Aref_ ## name, d_Bref_ ## name, d_Cref_ ## name, Ref::M, Ref::N, Ref::K); \
	hipError_t err_c_ ## name = hipGetLastError(); \
	if (err_c_ ## name != hipSuccess) { \
		std::cerr << "CUDA Error: " << hipGetErrorString(err_c_ ## name) << std::endl; \
	} \
	CUDA_CHECK(hipMemcpy(refC, d_Cref_ ## name, Ref::M * Ref::N * sizeof(float), hipMemcpyDeviceToHost)); \
	if (!ref.checkRef(refC)){ \
		std::cerr << "check ref failed!" << std::endl; \
	};

#define TIME(name) \
	float *d_A_ ## name, *d_B_ ## name, *d_C_ ## name; \
	CUDA_CHECK(hipMalloc(&d_A_ ## name, M * K * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_B_ ## name, K * N * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_C_ ## name, M * N * sizeof(float))); \
	CUDA_CHECK(hipMemcpy(d_A_ ## name, A, M * K * sizeof(float), hipMemcpyHostToDevice)); \
	CUDA_CHECK(hipMemcpy(d_B_ ## name, B, K * N * sizeof(float), hipMemcpyHostToDevice)); \
	hipEvent_t start_ ## name, end_ ## name; \
	hipEventCreate(&start_ ## name); \
	hipEventCreate(&end_ ## name); \
	float* d_C_INI_ ## name = new float[M * N](); \
	for (int i = 0; i < M; i++) { \
		for (int j = 0; j < N; j++) { \
			d_C_INI_ ## name[i * N + j] = 0; \
		} \
	} \
	for (int i = 0; i < 2; i++) \
	{ \
		CUDA_CHECK(hipMemcpy(d_C_ ## name, d_C_INI_ ## name, M * N * sizeof(float), hipMemcpyHostToDevice)); \
		name(d_A_ ## name, d_B_ ## name, d_C_ ## name, M, N, K); \
	} \
	hipError_t err_t_ ## name = hipGetLastError(); \
	if (err_t_ ## name != hipSuccess) { \
		std::cerr << "CUDA Error: " << hipGetErrorString(err_t_ ## name) << std::endl; \
	} \
	float milliseconds_ ## name = 0; \
	for (int i = 0; i < NUM_RUNS; i++) \
	{ \
		CUDA_CHECK(hipMemcpy(d_C_ ## name, d_C_INI_ ## name, M * N * sizeof(float), hipMemcpyHostToDevice)); \
		hipDeviceSynchronize(); \
		hipEventRecord(start_ ## name); \
		name(d_A_ ## name, d_B_ ## name, d_C_ ## name, M, N, K); \
		hipEventRecord(end_ ## name); \
		hipEventSynchronize(end_ ## name); \
		float milliseconds_ ## i = 0; \
		hipEventElapsedTime(&milliseconds_ ## i, start_ ## name, end_ ## name); \
		milliseconds_ ## name += milliseconds_ ## i; \
	} \
	hipMemcpy(C, d_C_ ## name, M * N * sizeof(float), hipMemcpyDeviceToHost); \
	std::cout << "Time taken for GEMM (GPU, " << #name <<"): " << milliseconds_ ## name / (float)NUM_RUNS << "ms" << std::endl; \
	hipFree(d_A_ ## name); \
	hipFree(d_B_ ## name); \
	hipFree(d_C_ ## name);

__global__ void gemm_gpu_o0_kernel(float* A, float* B, float *C, int M, int N, int K) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		for (int i = 0; i < M; i++) {
			for (int j = 0; j < N; j++) {
				for (int k = 0; k < K; k++) {
					C[i * N + j]  += A[i * K + k]  * B[k * N + j];
				}
			}
		}
    }
}

void gemm_gpu_o0(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
	dim3 blockSize(1);
	dim3 gridSize(1);
	gemm_gpu_o0_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

// The scafolding for optimized GEMM implementations
__global__ void gemm_gpu_o1_kernel(float* A, float* B, float *C, int M, int N, int K) {
	int out_col_id = blockIdx.x * BLOCK_WIDTH + threadIdx.x;
	int out_row_id = blockIdx.y * BLOCK_WIDTH + threadIdx.y;

	if (out_row_id < M && out_col_id < N){
		float acc = 0.0;
		for (int k = 0; k < K; k++) {
			acc += A[out_row_id * K + k]  * B[k * N + out_col_id];
		}
		C[out_row_id * N + out_col_id] = acc;
	}
}
void gemm_gpu_o1(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
	dim3 blockSize(BLOCK_WIDTH,BLOCK_WIDTH);
	dim3 gridSize((N+BLOCK_WIDTH-1)/BLOCK_WIDTH, (M+BLOCK_WIDTH-1)/BLOCK_WIDTH);
	gemm_gpu_o1_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

__global__ void gemm_gpu_o2_kernel(float* A, float* B, float *C, int M, int N, int K) {

	
	int out_row_id = blockIdx.y * BLOCK_WIDTH + threadIdx.y;
	int out_col_id = blockIdx.x * BLOCK_WIDTH + threadIdx.x;

	__shared__ float left_tile[BLOCK_WIDTH][BLOCK_WIDTH];
	__shared__ float right_tile[BLOCK_WIDTH][BLOCK_WIDTH];

	float acc = 0.0;

	for (int k_start=0;k_start<K;k_start += BLOCK_WIDTH){
		// left_tile[threadIdx.y][threadIdx.x] = k_start + threadIdx.x < K ? A[out_row_id*K + (k_start + threadIdx.x)] : 0;
		left_tile[threadIdx.x][threadIdx.y] = k_start + threadIdx.y < K ? A[(blockIdx.y * BLOCK_WIDTH + threadIdx.x)*K + (k_start + threadIdx.y)] : 0;
		right_tile[threadIdx.y][threadIdx.x] = k_start + threadIdx.y < K ? B[(k_start + threadIdx.y )*N + out_col_id ] : 0;
		__syncthreads();

		for (int inner_k=0; inner_k< BLOCK_WIDTH; inner_k++){
			acc += left_tile[threadIdx.y][inner_k] * right_tile[inner_k][threadIdx.x];
		}
		__syncthreads();
	}

	if (out_col_id< N && out_row_id < M){
		C[out_row_id*N + out_col_id] = acc;
	}
}
void gemm_gpu_o2(float* A, float* B, float* C, int M, int N, int K)
{
	// // Init block and grid size
	dim3 blockSize(BLOCK_WIDTH,BLOCK_WIDTH);
	dim3 gridSize((N+BLOCK_WIDTH-1)/BLOCK_WIDTH, (M+BLOCK_WIDTH-1)/BLOCK_WIDTH);
	gemm_gpu_o1_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

__global__ void gemm_gpu_o3_kernel(float* A, float* B, float *C, int M, int N, int K) {
	int out_row_id = blockIdx.y * BLOCK_WIDTH_OPT + threadIdx.y;
	int out_col_id = blockIdx.x * BLOCK_WIDTH_OPT + threadIdx.x;

	__shared__ float left_tile[BLOCK_WIDTH_OPT][BLOCK_WIDTH_OPT];
	__shared__ float right_tile[BLOCK_WIDTH_OPT][BLOCK_WIDTH_OPT];

	float acc = 0.0;

	for (int k_start=0;k_start<K;k_start += BLOCK_WIDTH_OPT){
		// left_tile[threadIdx.y][threadIdx.x] = k_start + threadIdx.x < K ? A[out_row_id*K + (k_start + threadIdx.x)] : 0;
		left_tile[threadIdx.x][threadIdx.y] = k_start + threadIdx.y < K ? A[(blockIdx.y * BLOCK_WIDTH_OPT + threadIdx.x)*K + (k_start + threadIdx.y)] : 0;
		right_tile[threadIdx.y][threadIdx.x] = k_start + threadIdx.y < K ? B[(k_start + threadIdx.y )*N + out_col_id ] : 0;
		__syncthreads();

		for (int inner_k=0; inner_k< BLOCK_WIDTH_OPT; inner_k++){
			acc += left_tile[threadIdx.y][inner_k] * right_tile[inner_k][threadIdx.x];
		}
		__syncthreads();
	}

	if (out_col_id< N && out_row_id < M){
		C[out_row_id*N + out_col_id] = acc;
	}
}
void gemm_gpu_o3(float* A, float* B, float* C, int M, int N, int K)
{
	dim3 blockSize(BLOCK_WIDTH_OPT,BLOCK_WIDTH_OPT);
	dim3 gridSize((N+BLOCK_WIDTH_OPT-1)/BLOCK_WIDTH_OPT, (M+BLOCK_WIDTH_OPT-1)/BLOCK_WIDTH_OPT);
	gemm_gpu_o1_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}



int main(int argc, char* argv[]) {
	if (argc < 3) {
		std::cout << "Usage: mp1 <M> <N> <K>" << std::endl;
		return 1;
	}

	int M = atoi(argv[1]);
	int N = atoi(argv[2]);
	int K = atoi(argv[3]);

	// int runs = atoi(argv[3]);
	float* A = new float[M * K]();
	float* B = new float[K * N]();
	float* C = new float[M * N]();

	fillRandom(A, M * K);
	fillRandom(B, K * N);

	/// GPU Implementation
        // Check if implementation is correct
	auto ref = Ref();
	float* refC = new float[Ref::M * Ref::N]();
 	CHECK(gemm_gpu_o0)
	CHECK(gemm_gpu_o1)
	CHECK(gemm_gpu_o2)
	CHECK(gemm_gpu_o3)

	// Actual run
 	TIME(gemm_gpu_o0)
	TIME(gemm_gpu_o1)
	TIME(gemm_gpu_o2)
	TIME(gemm_gpu_o3)

	hipHostFree(A);
	hipHostFree(B);
	hipHostFree(C);

	delete[] A;
	delete[] B;
	delete[] C;

	return 0;
}